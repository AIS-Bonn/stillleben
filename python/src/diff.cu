#include "hip/hip_runtime.h"
// stillleben differentiation module CUDA kernels
// Author: Arul Periyasamy <arul.periyasamy@ais.uni-bonn.de>

#include "diff.h"
#include <THC/THC.h>

extern THCState *state;

__device__ void clamp(int & idx, int maxIdx) {
    idx = idx > 0 ? idx : 0;
    idx = idx < (maxIdx - 1) ? idx : (maxIdx - 1);
}

const int filterWidth=3;

__global__ void generateSobelValidMaskKernel(const int16_t *instanceIndices,
    const float *depthImage, bool *validMask,
    const int numRows, const int numCols, const int shBlockElements)
{
    // copy block data to shared memory
    extern __shared__ int16_t shMemorySobel[];

    // divide the common shared memory between individual variables 
    int16_t *shInstanceIndices = shMemorySobel;
    float *shDepthImage = (float*)&shInstanceIndices[shBlockElements];

    const int2 thread2DIdx = make_int2(blockIdx.x * blockDim.x + threadIdx.x,
        blockIdx.y * blockDim.y + threadIdx.y);

    int halfWidth = filterWidth / 2;

    if(thread2DIdx.x < numCols && thread2DIdx.y < numRows)
    {
        // check if the thread corresponds boundary pixel of the block
        // if yes, load pixels in the neighboring window
        // else, load only the corresponding pixel.
        if(threadIdx.x == 0 || threadIdx.y == 0 || threadIdx.x == blockDim.x-1 || threadIdx.y == blockDim.y-1)
        {
            // boundary block pixels
            for(int x=-halfWidth; x<=halfWidth; ++x)
            {
                for(int y=-halfWidth; y<=halfWidth; ++y)
                {
                    int sh1DIdx = ((threadIdx.y + halfWidth + y) * (blockDim.x + (2 * halfWidth))) + (threadIdx.x + halfWidth + x) ;

                    int ty = thread2DIdx.y + y;
                    int tx = thread2DIdx.x + x;

                    clamp(ty, numRows);
                    clamp(tx, numCols); 

                    int thread1DIdx =  ty * numCols + tx;

                    shInstanceIndices[sh1DIdx] = instanceIndices[thread1DIdx];
                    shDepthImage[sh1DIdx] = depthImage[thread1DIdx];
                }
            }
        }
        else
        {
            // interior block pixels
            int thread1DIdx = thread2DIdx.y * numCols + thread2DIdx.x;
            int sh1DIdx = ((threadIdx.y+halfWidth) * (blockDim.x + (2* halfWidth))) + (threadIdx.x+halfWidth);
            shInstanceIndices[sh1DIdx] = instanceIndices[thread1DIdx];
            shDepthImage[sh1DIdx] = depthImage[thread1DIdx];
        }
        __syncthreads();

        int  thread1DIdx = thread2DIdx.y * numCols + thread2DIdx.x;
        int sh1DIdx = ((threadIdx.y+halfWidth) * (blockDim.x + (2* halfWidth))) + (threadIdx.x+halfWidth);

        bool isPixelValid = 1;

        // shInstanceIndices[sh1DIdx] == 0 are background pixels
        // background pixels are not interesting
        if(shInstanceIndices[sh1DIdx] != 0)
        {
            int16_t currentIndex = shInstanceIndices[sh1DIdx];
            float currentDepth= shDepthImage[sh1DIdx];
            for(int x=-halfWidth; x<=halfWidth; ++x)
            {
                for(int y=-halfWidth; y<=halfWidth; ++y)
                {
                    int shWindow1DIdx = ((threadIdx.y + halfWidth + y) * (blockDim.x + (2* halfWidth))) + (threadIdx.x + halfWidth + x);
                    if((shInstanceIndices[shWindow1DIdx] != currentIndex) &&
                        (shInstanceIndices[shWindow1DIdx] != 0) &&
                        (shDepthImage[shWindow1DIdx] < currentDepth))
                    {
                        isPixelValid = 0;
                    }
                }
            }
        }

        // write if isPixelValid is 0
        if(isPixelValid == 0)
        {
            validMask[ thread1DIdx ] = 0;
        }

    } //if(thread2DIdx.x < numCols && thread1DIdx.y < numRows)
}

__global__ void dilateObjectMaskKernel(const bool *objectMask, const bool *sobelValidMask, const float3 *coordinates,
    bool *dilatedMask, float3 *dilatedCoordinates, const int numRows, const int numCols, const int shBlockElements)
{
    // copy block data to shared memory
    extern __shared__ bool shMemoryDilate[];

    bool *shObjectMask = shMemoryDilate;
    bool *shSobelValidMask = (bool*)&shObjectMask[shBlockElements];
    float3 *shCoordinates = (float3*)&shSobelValidMask[shBlockElements];

    int halfWidth = filterWidth / 2;

    const int2 thread2DIdx = make_int2(blockIdx.x * blockDim.x + threadIdx.x,
        blockIdx.y * blockDim.y + threadIdx.y);

    // check if the thread corresponds boundary pixel of the block
    // if yes, load pixels in the neighboring window
    // else, load only the corresponding pixel.

    if(thread2DIdx.x < numCols && thread2DIdx.y < numRows)
    {
        if(threadIdx.x == 0 || threadIdx.y == 0 || threadIdx.x == blockDim.x-1 || threadIdx.y == blockDim.y-1)
        {
            // boundary block pixels
            for(int x=-halfWidth; x<=halfWidth; ++x)
            {
                for(int y=-halfWidth; y<=halfWidth; ++y)
                {
                    int sh1DIdx = ((threadIdx.y + halfWidth + y) * (blockDim.x + (2 * halfWidth))) + (threadIdx.x  + halfWidth + x);
                    
                    int ty = thread2DIdx.y + y;
                    int tx = thread2DIdx.x + x;
                    clamp(ty, numRows);
                    clamp(tx, numCols); 
                    int thread1DIdx =  ty * numCols + tx;

                    shObjectMask[sh1DIdx] = objectMask[thread1DIdx];
                    shSobelValidMask[sh1DIdx] = sobelValidMask[thread1DIdx];
                    shCoordinates[sh1DIdx] = coordinates[thread1DIdx];
                }
            }
        }
        else
        {   // interior block pixels
            int thread1DIdx = thread2DIdx.y * numCols + thread2DIdx.x;
            int sh1DIdx = ((threadIdx.y+halfWidth) * (blockDim.x + (2 * halfWidth))) + (threadIdx.x+halfWidth);
            shObjectMask[sh1DIdx] = objectMask[thread1DIdx];
            shSobelValidMask[sh1DIdx] = sobelValidMask[thread1DIdx];
            shCoordinates[sh1DIdx] = coordinates[thread1DIdx];
        }

        __syncthreads();

        int thread1DIdx = thread2DIdx.y * numCols + thread2DIdx.x;
        int sh1DIdx = ((threadIdx.y+halfWidth) * (blockDim.x + (2 * halfWidth))) + (threadIdx.x+halfWidth);

        bool outputMask = shObjectMask[sh1DIdx];
        float3 outputCoords = shCoordinates[sh1DIdx];

        if(outputMask == 0)
        {
            bool allValid = true;
            bool allBackground = true;
            for(int x=-halfWidth; x<=halfWidth; ++x)
            {
                for(int y=-halfWidth; y<=halfWidth; ++y)
                {
                    int shWindow1DIdx = ((threadIdx.y + halfWidth + y) * (blockDim.x + (2 * halfWidth))) + (threadIdx.x + halfWidth + x);
                    if(shObjectMask[shWindow1DIdx] != 0)
                    {
                        allBackground = false;
                        outputCoords = shCoordinates[shWindow1DIdx];
                    }

                    if(shSobelValidMask[shWindow1DIdx] == 0)
                    {
                        allValid = false;
                        break;
                    }
                }
            }

            // write dilated mask and coordinates for valid pixels
            if(!allBackground && allValid)
            {
                outputMask = 1;
            }
        } // if(shObjectMask[sh1DIdx] != 0)

        dilatedMask[thread1DIdx] = outputMask;
        dilatedCoordinates[thread1DIdx] = outputCoords;
    } // if(thread2DIdx.x < numCols && thread1DIdx.y < numRows)
}

namespace diff
{
    void generateSobelValidMaskCuda(torch::Tensor& instanceIndices,
        torch::Tensor& depthImage, torch::Tensor validMask)
    {
        // Type and shape checks are already done in bridge.cpp
        // skip sanity checks
        int16_t *instanceIndicesPtr = instanceIndices.data_ptr<int16_t>();
        float *depthImagePtr = depthImage.data_ptr<float>();
        bool *validMaskPtr = validMask.data_ptr<bool>();
        auto numRows = instanceIndices.size(0);
        auto numCols = instanceIndices.size(1);

        int threadsX = 32;
        int threadsY = 32;
        int sharedX = threadsX + 2 * (filterWidth / 2);
        int sharedY = threadsY + 2 * (filterWidth / 2);
        auto blocksX  = 1 + ((numCols - 1) / threadsX);
        auto blocksY  = 1 + ((numRows - 1) / threadsY);
        const dim3 blockSize(threadsX, threadsY);
        const dim3 gradSize(blocksX, blocksY);

        // const dim3 blockSize(1, 1);
        // const dim3 gradSize(1, 1);

        int shBlockElements = sharedX * sharedY;

        // compute total share memory per block needed
        int64_t shTotalMemory;
        shTotalMemory = sizeof(/* instanceIndices */ int16_t) * shBlockElements + sizeof (/* depthImage */ float) * shBlockElements;
        generateSobelValidMaskKernel<<<gradSize, blockSize, shTotalMemory>>> (instanceIndicesPtr, depthImagePtr, validMaskPtr, numRows, numCols, shBlockElements);
        hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
    } // void generateSobelValidMaskCuda

    void dilateObjectMaskCuda(torch::Tensor& objectMask,
        torch::Tensor& sobelValidMask, torch::Tensor& coordinates,
        torch::Tensor& dilatedMask, torch::Tensor& dilatedCoordinates)
    {
        // Type and shape checks are already done in bridge.cpp
        // skip sanity checks
        bool *objectMaskPtr = objectMask.data_ptr<bool>();
        bool *sobelValidMaskPtr = sobelValidMask.data_ptr<bool>();
        float *coordsPtr = coordinates.data_ptr<float>();
        bool *dilatedMaskPtr = dilatedMask.data_ptr<bool>();
        float *dilatedCoordsPtr = dilatedCoordinates.data_ptr<float>();

        float3 *coordinatesPtr = (float3 *)coordsPtr;
        float3 *dilatedCoordinatesPtr = (float3 *)dilatedCoordsPtr;

        int numRows = objectMask.size(0);
        int numCols = objectMask.size(1);

        int threadsX = 32;
        int threadsY = 32;
        int sharedX = threadsX + 2 * (filterWidth / 2);
        int sharedY = threadsY + 2 * (filterWidth / 2);
        int blocksX  = 1 + ((numCols - 1) / threadsX);
        int blocksY  = 1 + ((numRows - 1) / threadsY);
        const dim3 blockSize(threadsX, threadsY);
        const dim3 gradSize(blocksX, blocksY);

        // const dim3 blockSize(1, 1);
        // const dim3 gradSize(1, 1);

        int shBlockElements = sharedX * sharedY;

        // compute total share memory per block needed
        int64_t shTotalMemory;
        shTotalMemory = sizeof(/* objectMask */  bool) * shBlockElements + 
                        sizeof(/* sobelValidMask */ bool) * shBlockElements +
                        sizeof(/* coordsPtr */ float3) * shBlockElements;

        // shBlockElements can also be computed from blockDim.x, blockDim.y, and filterWidth insize each kernel thread
        // But, here we are doing it once and pass it as a param
        dilateObjectMaskKernel <<<gradSize, blockSize, shTotalMemory>>>
            (objectMaskPtr, sobelValidMaskPtr, coordinatesPtr, dilatedMaskPtr, dilatedCoordinatesPtr, numRows, numCols, shBlockElements);

        hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
    } // void dilateObjectMaskCuda
} //namespace diff
